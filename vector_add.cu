
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < 10) {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
    int a[10], b[10], c[10];
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, 10*sizeof(int));
    hipMalloc((void**)&dev_b, 10*sizeof(int));
    hipMalloc((void**)&dev_c, 10*sizeof(int));

    for (int i = 0; i < 10; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    hipMemcpy(dev_a, a, 10*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, 10*sizeof(int), hipMemcpyHostToDevice);

    add<<<10,1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, 10*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
